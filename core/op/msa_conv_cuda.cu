#include "hip/hip_runtime.h"
// Modify from https://github.com/open-mmlab/mmcv/blob/my_conv/mmcv/ops/csrc/common/cuda/deform_conv_cuda_kernel.cuh
// Copyright (c) OpenMMLab. All rights reserved.

#include "pytorch_cuda_helper.hpp"
#include <torch/torch.h>

template <typename T>
__global__ void msa_conv_im2col_gpu_kernel(
    const int n, const T *data_im, const int height,
    const int width, const int kernel_h, const int kernel_w, const int pad_h,
    const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int batch_size,
    const int num_channels, const int height_col,
    const int width_col, T *data_col)
{
    CUDA_1D_KERNEL_LOOP(index, n)
    {
        // index index of output matrix
        const int w_col = index % width_col;
        const int h_col = (index / width_col) % height_col;
        const int b_col = (index / width_col / height_col) % batch_size;
        const int c_im = (index / width_col / height_col) / batch_size;
        const int c_col = c_im * kernel_h * kernel_w;

        const int h_in = h_col * stride_h - pad_h;
        const int w_in = w_col * stride_w - pad_w;
        T *data_col_ptr =
            data_col +
            ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;
        const T *data_im_ptr =
            data_im + (b_col * num_channels + c_im) * height * width;

        for (int i = 0; i < kernel_h; ++i)
        {
            for (int j = 0; j < kernel_w; ++j)
            {
                T val = static_cast<T>(0);
                const int h_im = h_in + i * dilation_h;
                const int w_im = w_in + j * dilation_w;
                if (h_im > -1 && w_im > -1 && h_im < height && w_im < width)
                {
                    val = data_im_ptr[h_im * width + w_im];
                }
                *data_col_ptr = val;
                data_col_ptr = data_col_ptr + batch_size * height_col * width_col;
            }
        }
    }
}

void msa_conv_im2col_cuda(Tensor data_im,
                         const int channels, const int height,
                         const int width, const int ksize_h,
                         const int ksize_w, const int pad_h, const int pad_w,
                         const int stride_h, const int stride_w,
                         const int dilation_h, const int dilation_w,
                         const int parallel_imgs, Tensor data_col)
{
    int height_col =
        (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
    int width_col =
        (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
    int num_kernels = channels * height_col * width_col * parallel_imgs;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        data_im.scalar_type(), "msa_conv_im2col_gpu", [&]
        { msa_conv_im2col_gpu_kernel<scalar_t><<<GET_BLOCKS(num_kernels),
                                                THREADS_PER_BLOCK, 0,
                                                at::cuda::getCurrentCUDAStream()>>>(
              num_kernels, data_im.data_ptr<scalar_t>(),
              height, width, ksize_h, ksize_w,
              pad_h, pad_w, stride_h, stride_w,
              dilation_h, dilation_w,
              parallel_imgs, channels,
              height_col, width_col, data_col.data_ptr<scalar_t>()); });

    AT_CUDA_CHECK(hipGetLastError());
}